// Steps to run: 
// 1. nvcc .\simple_cuda.cu
// 2. .\a.exe


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_cuda() {
    printf("Hello from CUDA kernel!\n");
}

int main() {
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize(); // Wait for the kernel to finish
    std::cout << "Hello from CPU!" << std::endl;
    return 0;
}
