#include "hip/hip_runtime.h"
#include "elew_add.cuh"
#include <hip/hip_runtime.h>

__global__ void addKernel(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void elementwiseAdd(float* a, float* b, float* c, int n) {
    float *d_a, *d_b, *d_c;
    int size = n * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy input vectors from host to device memory
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host memory
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
